#include "hip/hip_runtime.h"
#include "simulation/particles.cuh"
#include "simulation/util.cuh"
#include <stdlib.h>
#include <cstddef>

// --- Utility Functions ---

void checkIfCopyPossible(ParticleSystem *ps_host, ParticleSystem *ps_device, unsigned int numParticles) {
	if (!ps_host || !ps_device) {
		fprintf(stderr, "Error: Invalid ParticleSystem pointers for copy operation.\n");
		exit(-100);
	}
	ParticleSystem ps_d_copy;
	CHECK_CUDA_ERROR(hipMemcpy(&ps_d_copy, ps_device, sizeof(ParticleSystem), hipMemcpyDeviceToHost));

	if (numParticles > ps_host->maxParticles || numParticles > ps_d_copy.maxParticles) {
		fprintf(stderr, "Error: Particle count (%u) must be smaller or equal to maxParticles.\n", numParticles);
		exit(-101);
	}
}

// --- Host Memory Functions ---

__host__ ParticleSystem *ParticleSystem_CreateOnHost(int maxParticles) {
	ParticleSystem *ps = (ParticleSystem *)malloc(sizeof(ParticleSystem));
	if (!ps) {
		fprintf(stderr, "Host memory allocation failed for ParticleSystem struct.\n");
		exit(-1);
	}

	ps->maxParticles = maxParticles;
	ps->numParticles = 0;

	if (ps->maxParticles > 0) {
		size_t numBytes_float4 = ps->maxParticles * sizeof(float4);
		size_t numBytes_float = ps->maxParticles * sizeof(float);

		ps->pos = (float4 *)malloc(numBytes_float4);
		ps->vel = (float4 *)malloc(numBytes_float4);
		ps->force = (float4 *)malloc(numBytes_float4);
		ps->mass = (float *)malloc(numBytes_float);
		ps->density = (float *)malloc(numBytes_float);
		ps->pressure = (float *)malloc(numBytes_float);
		ps->normal = (float4 *)malloc(numBytes_float4);
		ps->color_laplacian = (float *)malloc(numBytes_float);

		if (!ps->pos || !ps->vel || !ps->force || !ps->mass ||
		    !ps->density || !ps->pressure || !ps->normal || !ps->color_laplacian) {
			fprintf(stderr, "Host memory allocation failed for particle arrays.\n");
			exit(EXIT_FAILURE);
		}
	} else {
		ps->pos = nullptr;
		ps->vel = nullptr;
		ps->force = nullptr;
		ps->mass = nullptr;
		ps->density = nullptr;
		ps->pressure = nullptr;
		ps->normal = nullptr;
		ps->color_laplacian = nullptr;
	}
	return ps;
}

__host__ void ParticleSystem_FreeOnHost(ParticleSystem *ps) {
	if (ps) {
		if (ps->pos) free(ps->pos);
		if (ps->vel) free(ps->vel);
		if (ps->force) free(ps->force);
		if (ps->mass) free(ps->mass);
		if (ps->density) free(ps->density);
		if (ps->pressure) free(ps->pressure);
		if (ps->normal) free(ps->normal);
		if (ps->color_laplacian) free(ps->color_laplacian);
		free(ps);
	}
}

// --- Copy Host to Device Functions ---

__host__ void ParticleSystem_CopyAll_HostToDevice(ParticleSystem *ps_host, ParticleSystem *ps_device, unsigned int numParticles) {
	checkIfCopyPossible(ps_host, ps_device, numParticles);

	if (numParticles == 0) return;

	ParticleSystem ps_d_copy;
	CHECK_CUDA_ERROR(hipMemcpy(&ps_d_copy, ps_device, sizeof(ParticleSystem), hipMemcpyDeviceToHost));

	size_t numBytes_float4 = numParticles * sizeof(float4);
	size_t numBytes_float = numParticles * sizeof(float);

	CHECK_CUDA_ERROR(hipMemcpy(ps_d_copy.pos, ps_host->pos, numBytes_float4, hipMemcpyHostToDevice));
	CHECK_CUDA_ERROR(hipMemcpy(ps_d_copy.vel, ps_host->vel, numBytes_float4, hipMemcpyHostToDevice));
	CHECK_CUDA_ERROR(hipMemcpy(ps_d_copy.force, ps_host->force, numBytes_float4, hipMemcpyHostToDevice));
	CHECK_CUDA_ERROR(hipMemcpy(ps_d_copy.mass, ps_host->mass, numBytes_float, hipMemcpyHostToDevice));
	CHECK_CUDA_ERROR(hipMemcpy(ps_d_copy.density, ps_host->density, numBytes_float, hipMemcpyHostToDevice));
	CHECK_CUDA_ERROR(hipMemcpy(ps_d_copy.pressure, ps_host->pressure, numBytes_float, hipMemcpyHostToDevice));
	CHECK_CUDA_ERROR(hipMemcpy(ps_d_copy.normal, ps_host->normal, numBytes_float4, hipMemcpyHostToDevice));
	CHECK_CUDA_ERROR(hipMemcpy(ps_d_copy.color_laplacian, ps_host->color_laplacian, numBytes_float, hipMemcpyHostToDevice));
}

__host__ void ParticleSystem_CopyPos_HostToDevice(ParticleSystem *ps_host, ParticleSystem *ps_device, unsigned int numParticles) {
	checkIfCopyPossible(ps_host, ps_device, numParticles);
	if (numParticles == 0) return;
	ParticleSystem ps_d_copy;
	CHECK_CUDA_ERROR(hipMemcpy(&ps_d_copy, ps_device, sizeof(ParticleSystem), hipMemcpyDeviceToHost));
	CHECK_CUDA_ERROR(hipMemcpy(ps_d_copy.pos, ps_host->pos, numParticles * sizeof(float4), hipMemcpyHostToDevice));
}

__host__ void ParticleSystem_CopyVel_HostToDevice(ParticleSystem *ps_host, ParticleSystem *ps_device, unsigned int numParticles) {
	checkIfCopyPossible(ps_host, ps_device, numParticles);
	if (numParticles == 0) return;
	ParticleSystem ps_d_copy;
	CHECK_CUDA_ERROR(hipMemcpy(&ps_d_copy, ps_device, sizeof(ParticleSystem), hipMemcpyDeviceToHost));
	CHECK_CUDA_ERROR(hipMemcpy(ps_d_copy.vel, ps_host->vel, numParticles * sizeof(float4), hipMemcpyHostToDevice));
}

__host__ void ParticleSystem_CopyForce_HostToDevice(ParticleSystem *ps_host, ParticleSystem *ps_device, unsigned int numParticles) {
	checkIfCopyPossible(ps_host, ps_device, numParticles);
	if (numParticles == 0) return;
	ParticleSystem ps_d_copy;
	CHECK_CUDA_ERROR(hipMemcpy(&ps_d_copy, ps_device, sizeof(ParticleSystem), hipMemcpyDeviceToHost));
	CHECK_CUDA_ERROR(hipMemcpy(ps_d_copy.force, ps_host->force, numParticles * sizeof(float4), hipMemcpyHostToDevice));
}

__host__ void ParticleSystem_CopyMass_HostToDevice(ParticleSystem *ps_host, ParticleSystem *ps_device, unsigned int numParticles) {
	checkIfCopyPossible(ps_host, ps_device, numParticles);
	if (numParticles == 0) return;
	ParticleSystem ps_d_copy;
	CHECK_CUDA_ERROR(hipMemcpy(&ps_d_copy, ps_device, sizeof(ParticleSystem), hipMemcpyDeviceToHost));
	CHECK_CUDA_ERROR(hipMemcpy(ps_d_copy.mass, ps_host->mass, numParticles * sizeof(float), hipMemcpyHostToDevice));
}

__host__ void ParticleSystem_CopyDensity_HostToDevice(ParticleSystem *ps_host, ParticleSystem *ps_device, unsigned int numParticles) {
	checkIfCopyPossible(ps_host, ps_device, numParticles);
	if (numParticles == 0) return;
	ParticleSystem ps_d_copy;
	CHECK_CUDA_ERROR(hipMemcpy(&ps_d_copy, ps_device, sizeof(ParticleSystem), hipMemcpyDeviceToHost));
	CHECK_CUDA_ERROR(hipMemcpy(ps_d_copy.density, ps_host->density, numParticles * sizeof(float), hipMemcpyHostToDevice));
}

__host__ void ParticleSystem_CopyPressure_HostToDevice(ParticleSystem *ps_host, ParticleSystem *ps_device, unsigned int numParticles) {
	checkIfCopyPossible(ps_host, ps_device, numParticles);
	if (numParticles == 0) return;
	ParticleSystem ps_d_copy;
	CHECK_CUDA_ERROR(hipMemcpy(&ps_d_copy, ps_device, sizeof(ParticleSystem), hipMemcpyDeviceToHost));
	CHECK_CUDA_ERROR(hipMemcpy(ps_d_copy.pressure, ps_host->pressure, numParticles * sizeof(float), hipMemcpyHostToDevice));
}

__host__ void ParticleSystem_CopyNormal_HostToDevice(ParticleSystem *ps_host, ParticleSystem *ps_device, unsigned int numParticles) {
	checkIfCopyPossible(ps_host, ps_device, numParticles);
	if (numParticles == 0) return;
	ParticleSystem ps_d_copy;
	CHECK_CUDA_ERROR(hipMemcpy(&ps_d_copy, ps_device, sizeof(ParticleSystem), hipMemcpyDeviceToHost));
	CHECK_CUDA_ERROR(hipMemcpy(ps_d_copy.normal, ps_host->normal, numParticles * sizeof(float4), hipMemcpyHostToDevice));
}

__host__ void ParticleSystem_CopyColorLaplacian_HostToDevice(ParticleSystem *ps_host, ParticleSystem *ps_device, unsigned int numParticles) {
	checkIfCopyPossible(ps_host, ps_device, numParticles);
	if (numParticles == 0) return;
	ParticleSystem ps_d_copy;
	CHECK_CUDA_ERROR(hipMemcpy(&ps_d_copy, ps_device, sizeof(ParticleSystem), hipMemcpyDeviceToHost));
	CHECK_CUDA_ERROR(hipMemcpy(ps_d_copy.color_laplacian, ps_host->color_laplacian, numParticles * sizeof(float), hipMemcpyHostToDevice));
}

// --- Device Memory Functions ---

__host__ ParticleSystem *ParticleSystem_CreateOnDevice(int maxParticles) {
	ParticleSystem *d_ps;
	CHECK_CUDA_ERROR(hipMalloc((void **)&d_ps, sizeof(ParticleSystem)));

	// Create a temporary host-side struct to configure before copying to device
	ParticleSystem h_ps;

	h_ps.maxParticles = maxParticles;
	h_ps.numParticles = 0;

	if (maxParticles > 0) {
		size_t numBytes_float4 = maxParticles * sizeof(float4);
		size_t numBytes_float = maxParticles * sizeof(float);
		CHECK_CUDA_ERROR(hipMalloc((void **)&h_ps.pos, numBytes_float4));
		CHECK_CUDA_ERROR(hipMalloc((void **)&h_ps.vel, numBytes_float4));
		CHECK_CUDA_ERROR(hipMalloc((void **)&h_ps.force, numBytes_float4));
		CHECK_CUDA_ERROR(hipMalloc((void **)&h_ps.mass, numBytes_float));
		CHECK_CUDA_ERROR(hipMalloc((void **)&h_ps.density, numBytes_float));
		CHECK_CUDA_ERROR(hipMalloc((void **)&h_ps.pressure, numBytes_float));
		CHECK_CUDA_ERROR(hipMalloc((void **)&h_ps.normal, numBytes_float4));
		CHECK_CUDA_ERROR(hipMalloc((void **)&h_ps.color_laplacian, numBytes_float));
	}
	CHECK_CUDA_ERROR(hipMemcpy(d_ps, &h_ps, sizeof(ParticleSystem), hipMemcpyHostToDevice));
	return d_ps;
}

__host__ void ParticleSystem_FreeOnDevice(ParticleSystem *ps_device) {
	if (ps_device) {
		ParticleSystem ps;
		CHECK_CUDA_ERROR(hipMemcpy(&ps, ps_device, sizeof(ParticleSystem), hipMemcpyDeviceToHost));
		if (ps.maxParticles > 0) {
			CHECK_CUDA_ERROR(hipFree(ps.pos));
			CHECK_CUDA_ERROR(hipFree(ps.vel));
			CHECK_CUDA_ERROR(hipFree(ps.force));
			CHECK_CUDA_ERROR(hipFree(ps.mass));
			CHECK_CUDA_ERROR(hipFree(ps.density));
			CHECK_CUDA_ERROR(hipFree(ps.pressure));
			CHECK_CUDA_ERROR(hipFree(ps.normal));
			CHECK_CUDA_ERROR(hipFree(ps.color_laplacian));
		}
		CHECK_CUDA_ERROR(hipFree(ps_device));
	}
}

// --- Device Accessor Functions ---

__host__ void ParticleSystem_SetNumParticlesOnDevice(ParticleSystem *ps_device, int numParticles) {
	CHECK_CUDA_ERROR(hipMemcpy((char *)ps_device + offsetof(ParticleSystem, numParticles), &numParticles, sizeof(int), hipMemcpyHostToDevice));
}

__host__ unsigned int ParticleSystem_GetNumParticlesOnDevice(ParticleSystem *ps_device) {
	unsigned int numParticles;
	CHECK_CUDA_ERROR(hipMemcpy(&numParticles, (char *)ps_device + offsetof(ParticleSystem, numParticles), sizeof(unsigned int), hipMemcpyDeviceToHost));
	return numParticles;
}

// --- Copy Device to Host Functions ---

__host__ void ParticleSystem_CopyAll_DeviceToHost(ParticleSystem *ps_host, ParticleSystem *ps_device, unsigned int numParticles) {
	checkIfCopyPossible(ps_host, ps_device, numParticles);

	if (numParticles == 0) return;

	ParticleSystem ps_d_copy;
	CHECK_CUDA_ERROR(hipMemcpy(&ps_d_copy, ps_device, sizeof(ParticleSystem), hipMemcpyDeviceToHost));

	size_t numBytes_float4 = numParticles * sizeof(float4);
	size_t numBytes_float = numParticles * sizeof(float);

	CHECK_CUDA_ERROR(hipMemcpy(ps_host->pos, ps_d_copy.pos, numBytes_float4, hipMemcpyDeviceToHost));
	CHECK_CUDA_ERROR(hipMemcpy(ps_host->vel, ps_d_copy.vel, numBytes_float4, hipMemcpyDeviceToHost));
	CHECK_CUDA_ERROR(hipMemcpy(ps_host->force, ps_d_copy.force, numBytes_float4, hipMemcpyDeviceToHost));
	CHECK_CUDA_ERROR(hipMemcpy(ps_host->mass, ps_d_copy.mass, numBytes_float, hipMemcpyDeviceToHost));
	CHECK_CUDA_ERROR(hipMemcpy(ps_host->density, ps_d_copy.density, numBytes_float, hipMemcpyDeviceToHost));
	CHECK_CUDA_ERROR(hipMemcpy(ps_host->pressure, ps_d_copy.pressure, numBytes_float, hipMemcpyDeviceToHost));
	CHECK_CUDA_ERROR(hipMemcpy(ps_host->normal, ps_d_copy.normal, numBytes_float4, hipMemcpyDeviceToHost));
	CHECK_CUDA_ERROR(hipMemcpy(ps_host->color_laplacian, ps_d_copy.color_laplacian, numBytes_float, hipMemcpyDeviceToHost));
}

__host__ void ParticleSystem_CopyPos_DeviceToHost(ParticleSystem *ps_host, ParticleSystem *ps_device, unsigned int numParticles) {
	checkIfCopyPossible(ps_host, ps_device, numParticles);
	if (numParticles == 0) return;
	ParticleSystem ps_d_copy;
	CHECK_CUDA_ERROR(hipMemcpy(&ps_d_copy, ps_device, sizeof(ParticleSystem), hipMemcpyDeviceToHost));
	CHECK_CUDA_ERROR(hipMemcpy(ps_host->pos, ps_d_copy.pos, numParticles * sizeof(float4), hipMemcpyDeviceToHost));
}

__host__ void ParticleSystem_CopyVel_DeviceToHost(ParticleSystem *ps_host, ParticleSystem *ps_device, unsigned int numParticles) {
	checkIfCopyPossible(ps_host, ps_device, numParticles);
	if (numParticles == 0) return;
	ParticleSystem ps_d_copy;
	CHECK_CUDA_ERROR(hipMemcpy(&ps_d_copy, ps_device, sizeof(ParticleSystem), hipMemcpyDeviceToHost));
	CHECK_CUDA_ERROR(hipMemcpy(ps_host->vel, ps_d_copy.vel, numParticles * sizeof(float4), hipMemcpyDeviceToHost));
}

__host__ void ParticleSystem_CopyForce_DeviceToHost(ParticleSystem *ps_host, ParticleSystem *ps_device, unsigned int numParticles) {
	checkIfCopyPossible(ps_host, ps_device, numParticles);
	if (numParticles == 0) return;
	ParticleSystem ps_d_copy;
	CHECK_CUDA_ERROR(hipMemcpy(&ps_d_copy, ps_device, sizeof(ParticleSystem), hipMemcpyDeviceToHost));
	CHECK_CUDA_ERROR(hipMemcpy(ps_host->force, ps_d_copy.force, numParticles * sizeof(float4), hipMemcpyDeviceToHost));
}

__host__ void ParticleSystem_CopyMass_DeviceToHost(ParticleSystem *ps_host, ParticleSystem *ps_device, unsigned int numParticles) {
	checkIfCopyPossible(ps_host, ps_device, numParticles);
	if (numParticles == 0) return;
	ParticleSystem ps_d_copy;
	CHECK_CUDA_ERROR(hipMemcpy(&ps_d_copy, ps_device, sizeof(ParticleSystem), hipMemcpyDeviceToHost));
	CHECK_CUDA_ERROR(hipMemcpy(ps_host->mass, ps_d_copy.mass, numParticles * sizeof(float), hipMemcpyDeviceToHost));
}

__host__ void ParticleSystem_CopyDensity_DeviceToHost(ParticleSystem *ps_host, ParticleSystem *ps_device, unsigned int numParticles) {
	checkIfCopyPossible(ps_host, ps_device, numParticles);
	if (numParticles == 0) return;
	ParticleSystem ps_d_copy;
	CHECK_CUDA_ERROR(hipMemcpy(&ps_d_copy, ps_device, sizeof(ParticleSystem), hipMemcpyDeviceToHost));
	CHECK_CUDA_ERROR(hipMemcpy(ps_host->density, ps_d_copy.density, numParticles * sizeof(float), hipMemcpyDeviceToHost));
}

__host__ void ParticleSystem_CopyPressure_DeviceToHost(ParticleSystem *ps_host, ParticleSystem *ps_device, unsigned int numParticles) {
	checkIfCopyPossible(ps_host, ps_device, numParticles);
	if (numParticles == 0) return;
	ParticleSystem ps_d_copy;
	CHECK_CUDA_ERROR(hipMemcpy(&ps_d_copy, ps_device, sizeof(ParticleSystem), hipMemcpyDeviceToHost));
	CHECK_CUDA_ERROR(hipMemcpy(ps_host->pressure, ps_d_copy.pressure, numParticles * sizeof(float), hipMemcpyDeviceToHost));
}

__host__ void ParticleSystem_CopyNormal_DeviceToHost(ParticleSystem *ps_host, ParticleSystem *ps_device, unsigned int numParticles) {
	checkIfCopyPossible(ps_host, ps_device, numParticles);
	if (numParticles == 0) return;
	ParticleSystem ps_d_copy;
	CHECK_CUDA_ERROR(hipMemcpy(&ps_d_copy, ps_device, sizeof(ParticleSystem), hipMemcpyDeviceToHost));
	CHECK_CUDA_ERROR(hipMemcpy(ps_host->normal, ps_d_copy.normal, numParticles * sizeof(float4), hipMemcpyDeviceToHost));
}

__host__ void ParticleSystem_CopyColorLaplacian_DeviceToHost(ParticleSystem *ps_host, ParticleSystem *ps_device, unsigned int numParticles) {
	checkIfCopyPossible(ps_host, ps_device, numParticles);
	if (numParticles == 0) return;
	ParticleSystem ps_d_copy;
	CHECK_CUDA_ERROR(hipMemcpy(&ps_d_copy, ps_device, sizeof(ParticleSystem), hipMemcpyDeviceToHost));
	CHECK_CUDA_ERROR(hipMemcpy(ps_host->color_laplacian, ps_d_copy.color_laplacian, numParticles * sizeof(float), hipMemcpyDeviceToHost));
}