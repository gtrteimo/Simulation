#include "cuda/util.cuh"


int main(void) {
	// Initialize CUDA
	hipError_t err = hipSetDevice(0);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to set device: %s\n", hipGetErrorString(err));
		return -1;
	}

	// Insert Simulation Code Here

	// Clean up and exit
	hipDeviceReset();
	return 0;
}